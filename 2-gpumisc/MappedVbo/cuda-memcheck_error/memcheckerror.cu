#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GL/glut.h>
#include <iostream>
#include <cuda_gl_interop.h>

using namespace std;

#define BLOCK_SIZE 128
__global__ void simpleKernel(
        float* output )
{
    output[threadIdx.x] = 0;
}

hipGraphicsResource* positionsVBO_CUDA;

void display()
{
    hipError_t cuda_inited = cudaGLSetGLDevice(0);
    int glew_inited = glewInit();

    unsigned N = BLOCK_SIZE;
    unsigned size = N*sizeof(float);
    unsigned vbo;
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    hipError_t is_registered = hipGraphicsGLRegisterBuffer( &positionsVBO_CUDA, vbo, cudaGraphicsMapFlagsWriteDiscard);
    hipError_t is_mapped = hipGraphicsMapResources(1, &positionsVBO_CUDA, 0 );
    float* g_data;
    size_t num_bytes;
    hipError_t got_pointer = hipGraphicsResourceGetMappedPointer((void**)&g_data, &num_bytes, positionsVBO_CUDA);

    dim3 block( BLOCK_SIZE );
    dim3 grid( 1 );
    simpleKernel<<< grid, block>>>(g_data);

    hipError_t unmapped = hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
    hipError_t unreg = hipGraphicsUnregisterResource( positionsVBO_CUDA );

    cout << "cuda_inited = " << (cuda_inited == hipSuccess) << endl;
    cout << "glew_inited = " << (glew_inited == 0) << endl;
    cout << "is_registered = "<< (is_registered == hipSuccess) << endl;
    cout << "is_mapped = "<< (is_mapped == hipSuccess) << endl;
    cout << "num_bytes = " << num_bytes << endl;
    cout << "g_data = " << g_data << endl;
    cout << "got_pointer = "<< (got_pointer == hipSuccess) << endl;
    cout << "unmapped = "<< (unmapped == hipSuccess) << endl;
    cout << "unreg = "<< (unreg == hipSuccess) << endl;

    ::exit(0);
}


int main(int argc, char *argv[])
{
    glutInit(&argc,argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(500, 500);
    glutInitWindowPosition(300, 200);
    glutCreateWindow("Mapped VBO test");
    glutDisplayFunc( display );
    glutMainLoop();
    return 0;
}
