#include "hip/hip_runtime.h"
#include "gl.h"
#ifndef __APPLE__
#   include <GL/glut.h>
#else
#   include <GLUT/glut.h>
#endif
#include <iostream>
#include <fstream>
#include <cuda_gl_interop.h>

using namespace std;

#define BLOCK_SIZE 128
__global__ void simpleKernel(
        float* output )
{
    output[threadIdx.x] = 0;
}

hipGraphicsResource* positionsVBO_CUDA;

void display()
{
    glClear(GL_COLOR_BUFFER_BIT);
	glutSwapBuffers();
	
    static bool once = true;
	if (!once)
	    return;
    once = false;
	
    hipError_t cuda_inited = cudaGLSetGLDevice(0);
#ifndef __APPLE__ // glewInit is not needed on Mac
    int glew_inited = glewInit();
#endif

    unsigned N = BLOCK_SIZE;
    unsigned size = N*sizeof(float);
    unsigned vbo;
    glGenBuffers(1, &vbo);
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    glBufferData(GL_ARRAY_BUFFER, size, 0, GL_DYNAMIC_DRAW);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
    hipError_t is_registered = hipGraphicsGLRegisterBuffer( &positionsVBO_CUDA, vbo, cudaGraphicsMapFlagsWriteDiscard);
    hipError_t is_mapped = hipGraphicsMapResources(1, &positionsVBO_CUDA, 0 );
    float* g_data;
    size_t num_bytes;
    hipError_t got_pointer = hipGraphicsResourceGetMappedPointer((void**)&g_data, &num_bytes, positionsVBO_CUDA);

    float* g_data2;
    hipError_t mallocd = hipMalloc( &g_data2, size );
    hipError_t memcopied = hipMemcpy( g_data2, g_data, size, hipMemcpyDeviceToDevice );

    dim3 block( BLOCK_SIZE );
    dim3 grid( 1 );
    simpleKernel<<< grid, block>>>(g_data2);
    hipError_t freed = hipFree( g_data2 );

    hipError_t unmapped = hipGraphicsUnmapResources(1, &positionsVBO_CUDA, 0);
    hipError_t unreg = hipGraphicsUnregisterResource( positionsVBO_CUDA );
	hipError_t sync = hipDeviceSynchronize();

    bool all_success = (cuda_inited == hipSuccess)
#ifndef __APPLE__ // glewInit is not needed on Mac
        && (glew_inited == 0)
#endif
        && (is_registered == hipSuccess)
        && (is_mapped == hipSuccess)
        && (size == num_bytes)
        && (0 != g_data)
        && (got_pointer == hipSuccess)
		&& (mallocd == hipSuccess)
		&& (memcopied == hipSuccess)
		&& (freed == hipSuccess)
        && (unmapped == hipSuccess)
        && (unmapped == hipSuccess)
        && (unreg == hipSuccess)
        && (sync == hipSuccess);

    cout<< "all_success = " << all_success << endl
        << "cuda_inited = " << (cuda_inited == hipSuccess) << endl
#ifndef __APPLE__ // glewInit is not needed on Mac
        << "glew_inited = " << (glew_inited == 0) << endl
#endif
        << "is_registered = "<< (is_registered == hipSuccess) << endl
        << "is_mapped = "<< (is_mapped == hipSuccess) << endl
        << "num_bytes = " << num_bytes << endl
        << "size = " << size << endl
        << "g_data = " << g_data << endl
        << "got_pointer = " << (got_pointer == hipSuccess) << endl
		<< "mallocd = " << (mallocd == hipSuccess) << endl
		<< "memcopied = " << (memcopied == hipSuccess) << endl
		<< "freed = " << (freed == hipSuccess) << endl
        << "unmapped = "<< (unmapped == hipSuccess) << endl
        << "unreg = "<< (unreg == hipSuccess) << endl
        << "sync = "<< (sync == hipSuccess) << endl;
	
    bool any_failed = !all_success;
    exit(any_failed);
}


int main(int argc, char *argv[])
{
    glutInit(&argc,argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(500, 500);
    glutInitWindowPosition(300, 200);
    glutCreateWindow(__FILE__);
    glutDisplayFunc( display );
    glutMainLoop();
    return 0;
}
