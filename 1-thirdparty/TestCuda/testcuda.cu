#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

using namespace std;

#define BLOCK_SIZE 128
__global__ void simpleKernel(
        float* output )
{
    output[threadIdx.x] = 0;
}


int main(int argc, char *argv[])
{
    unsigned N = BLOCK_SIZE;
    unsigned size = N*sizeof(float);
    float* g_data;
    hipError_t mallocd = hipMalloc( &g_data, size );
    dim3 block( BLOCK_SIZE );
    dim3 grid( 1 );
    simpleKernel<<< grid, block>>>(g_data);
    hipError_t freed = hipFree( g_data );

	hipError_t sync = hipDeviceSynchronize();

    bool all_success = (mallocd == hipSuccess)
		&& (freed == hipSuccess)
        && (sync == hipSuccess);

    cout << "mallocd = " << (mallocd == hipSuccess) << endl
         << "freed = " << (freed == hipSuccess) << endl
         << "sync = " << (sync == hipSuccess) << endl;
	
    bool any_failed = !all_success;
    exit(any_failed);
}
