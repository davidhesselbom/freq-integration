#include "hip/hip_runtime.h"
#ifndef __APPLE__
#   include <GL/glut.h>
#else
#   include <GLUT/glut.h>
#endif
#include <iostream>
#include <fstream>

using namespace std;

#define BLOCK_SIZE 128
__global__ void simpleKernel(
        float* output )
{
    output[threadIdx.x] = 0;
}

hipGraphicsResource* positionsVBO_CUDA;

void display()
{
    glClear(GL_COLOR_BUFFER_BIT);
	glutSwapBuffers();
	
    static bool once = true;
	if (!once)
	    return;
    once = false;
	
    unsigned N = BLOCK_SIZE;
    unsigned size = N*sizeof(float);
    float* g_data;
    hipError_t mallocd = hipMalloc( &g_data, size );
    dim3 block( BLOCK_SIZE );
    dim3 grid( 1 );
    simpleKernel<<< grid, block>>>(g_data);
	
    hipError_t freed = hipFree( g_data );
	hipError_t sync = hipDeviceSynchronize();

    bool all_success = (mallocd == hipSuccess)
		&& (freed == hipSuccess)
        && (sync == hipSuccess);

    cout << "all_success = " << all_success << endl
         << "mallocd = " << (mallocd == hipSuccess) << endl
         << "freed = " << (freed == hipSuccess) << endl
         << "sync = " << (sync == 0) << endl;
	
    bool any_failed = !all_success;
    exit(any_failed);
}


int main(int argc, char *argv[])
{
    glutInit(&argc,argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_DOUBLE | GLUT_DEPTH);
    glutInitWindowSize(500, 500);
    glutInitWindowPosition(300, 200);
    glutCreateWindow(__FILE__);
    glutDisplayFunc( display );
    glutMainLoop();
    return 0;
}
